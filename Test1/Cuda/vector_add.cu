
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

#define N 100000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;

    // Allocate memory
    if ((a = (float*)malloc(sizeof(float) * N)) == NULL) {
	    printf("a not allocated\n");
    };
    if ((b = (float*)malloc(sizeof(float) * N)) == NULL) {
	    printf("b not allocated\n");
    }
    if ((out = (float*)malloc(sizeof(float) * N)) == NULL) {
	    printf("out not allocated\n");
    };

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }
    hipMalloc((void **) & d_a, sizeof(float)*N);
    printf("d_a = %p\n", (void **) &d_a);
    hipMalloc((void **) & d_b, sizeof(float)*N);
    printf("d_b = %p\n", (void **) &d_b);
    hipMalloc((void **) & d_out, sizeof(float)*N);
    printf("d_out = %p\n", (void **) &d_out);

    hipMemcpy(d_a,a,sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,sizeof(float)*N, hipMemcpyHostToDevice);
    // Main function
    vector_add<<<1,1>>> (d_out, d_a, d_b, N);
    hipMemcpy(out,d_out,sizeof(float)*N, hipMemcpyDeviceToHost);
    int i = rand() %100000;
    printf("out[%d]=%f\n",i,out[i]);
    assert(out[i] == 3);
}
