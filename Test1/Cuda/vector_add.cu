
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 100000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;

    // Allocate memory
    if ((a = (float*)malloc(sizeof(float) * N)) == NULL) {
	    printf("a not allocated\n");
    };
    if ((b = (float*)malloc(sizeof(float) * N)) == NULL) {
	    printf("b not allocated\n");
    }
    if ((out = (float*)malloc(sizeof(float) * N)) == NULL) {
	    printf("out not allocated\n");
    };

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }
    hipMalloc((void **) & d_a, sizeof(float)*N);
    hipMalloc((void **) & d_b, sizeof(float)*N);
    hipMalloc((void **) & d_out, sizeof(float)*N);

    hipMemcpy(d_a,a,sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,sizeof(float)*N, hipMemcpyHostToDevice);
    // Main function
    vector_add<<<1,1>>> (d_out, d_a, d_b, N);
    hipMemcpy(out,d_out,sizeof(float)*N, hipMemcpyDeviceToHost);
}
