
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>

#define N 10000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 
    float *d_a, *d_b, *d_out;

    // Allocate memory
    if ((a = (float*)malloc(sizeof(float) * N)) == NULL) {
	    printf("a not allocated\n");
    };
    if ((b = (float*)malloc(sizeof(float) * N)) == NULL) {
	    printf("b not allocated\n");
    }
    if ((out = (float*)malloc(sizeof(float) * N)) == NULL) {
	    printf("out not allocated\n");
    };

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }
    hipMalloc((void **) & d_a, sizeof(float)*N);
    printf("d_a = %p\n", (void **) &d_a);
    hipMalloc((void **) & d_b, sizeof(float)*N);
    printf("d_b = %p\n", (void **) &d_b);
    hipMalloc((void **) & d_out, sizeof(float)*N);
    printf("d_out = %p\n", (void **) &d_out);

    hipMemcpy(d_a,a,sizeof(float)*N, hipMemcpyHostToDevice);
    printf("a -> d_a\n");
    hipMemcpy(d_b,b,sizeof(float)*N, hipMemcpyHostToDevice);
    printf("b -> d_b\n");
    // Main function
    vector_add<<<1,1>>> (d_out, d_a, d_b, N);
    hipMemcpy(out,d_out,sizeof(float)*N, hipMemcpyDeviceToHost);
    printf("d_out -> out\n");
    int i = rand() %N;
    printf("out[%d]=%f\n",i,out[i]);
    assert(out[i] == 3);
}
